#include "hip/hip_runtime.h"
// v0：使用共享内存 SmemSoftmax latency = 28.892096 ms
// v1: 向量化的读写

# include <hip/hip_runtime.h>
# include <hip/hip_runtime.h>
# include <iostream>

// 并行性：每个 block 处理 1024 个数据，每个 block 独享一个shared memory
// 选择 block 中 tid=0 的线程来求 max_val 和 sum
__global__ void softmax_v1(float* x, float* y, int n){
    int tid = threadIdx.x;      // 256
    int gtid = blockDim.x * blockIdx.x + threadIdx.x;

    // 加载数据到 shared memory 空间
    __shared__ float smem[1024];
    smem[tid * 4 + 0] = x[gtid * 4 + 0];
    smem[tid * 4 + 1] = x[gtid * 4 + 1];
    smem[tid * 4 + 2] = x[gtid * 4 + 2];
    smem[tid * 4 + 3] = x[gtid * 4 + 3];
    __syncthreads();

    // 每个 block 内求最大值
    __shared__ float max_val;
    max_val = 0;
    if (tid == 0){
        for (int i = 0; i < 1024; ++i){
            max_val = max(max_val, smem[i]);
        }
    }
    __syncthreads();

    // 每个 block 内求 sum
    __shared__ float sum;
    sum = 0;
    if (tid == 0){
        for (int i = 0; i < 1024; ++i){
            sum += exp(smem[i] - max_val);
        }
    }
    __syncthreads();

    y[gtid * 4 + 0] = exp(smem[tid * 4 + 0] - max_val) / sum;
    y[gtid * 4 + 1] = exp(smem[tid * 4 + 1] - max_val) / sum;
    y[gtid * 4 + 2] = exp(smem[tid * 4 + 2] - max_val) / sum;
    y[gtid * 4 + 3] = exp(smem[tid * 4 + 3] - max_val) / sum;

    __syncthreads();
}

// int N = 1000 * 1024
// softmax公式
// e^(xi - max(xi)) / sigma(e^(xi - max(xi)))
void softmaxCPU(float* input, float* groundtruth, int rows, int cols){
    for (int j = 0; j < rows; ++j){
        float total = 0;
        float MAX = 0;
        for (int i = 0; i < cols; ++i){
            MAX = max(input[j * cols + i], MAX);          // 找最大值 max(x1)
        }
        for (int i = 0; i < cols; ++i){
            total += exp(input[j * cols + i] - MAX);      // 防止softmax溢出
        }
        for (int i = 0; i < cols; ++i){
            groundtruth[j * cols + i] = exp(input[j * cols + i] - MAX) / total;
        }
    }
}
void checkResult(float* out, float* groundtruth, int n){
    for (int i = 0; i < n; i += 1024){
        if (out[i] - groundtruth[i] > 1e-5){
            printf("the ans is false");
            printf(" the out is: %f", out[i]);
            printf(" the groundtruth is: %f\n", groundtruth[i]);
        }else{
            printf("the ans is true!");
            printf(" the out is: %f", out[i]);
            printf(" the groundtruth is: %f\n", groundtruth[i]);
        }
    }
}

int main(){
    const int N = 1000 * 1024;

    float* x_host = (float* )malloc(N * sizeof(float));
    float* y_host = (float* )malloc(N * sizeof(float));
    float* groundtruth = (float* )malloc(N * sizeof(float));
    float* x_device;
    float* y_device;
    hipMalloc((void** )&x_device, N * sizeof(float));
    hipMalloc((void** )&y_device, N * sizeof(float));

    // 初始化
    for (int i = 0; i < N; ++i){
        x_host[i] = i % 10;
    }
    hipMemcpy(x_device, x_host, N * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;    // 1024 / 4 = 256
    int gridSize = (N + 1024 - 1) / 1024;
    dim3 block(blockSize);
    dim3 grid(gridSize);

    float milliseconds = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    softmax_v1<<<grid, block>>>(x_device, y_device, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(y_host, y_device, N * sizeof(float), hipMemcpyDeviceToHost);

    softmaxCPU(x_host, groundtruth, 1000, 1024);
    checkResult(y_host, groundtruth, N);

    printf("SmemSoftmax latency = %f ms\n", milliseconds);

    hipFree(x_device);
    hipFree(y_device);
    free(x_host);
    free(y_host);
    free(groundtruth);
}