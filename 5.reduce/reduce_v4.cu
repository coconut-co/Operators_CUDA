// 累加
// baseline                              reduce_baseline latency = 503.954346 ms
// 引入shared memory且并行化处理           reduce_v0 latency = 0.719744 ms    
// 手动选择线程                            reduce_v1 latency = 0.503008 ms
// 用位运算来代替除余操作和除法操作         reduce_v1 latency = 0.509184 ms
// 消除shared memory 的 bank conflict     reduce_v2 latency = 0.478688 ms
// 减少空闲线程                            reduce_v3 latency = 0.269568 ms
// 展开for循环最后一个warp，只有warp0工作   reduce_v4 latency = 0.220992 ms

# include<iostream>
# include<hip/hip_runtime.h>


//__device__：GPU端的函数，由编译器决定是否inline
__device__ void warpSharedMemReduce(volatile float* smem, int tid){
    // 自从volat架构后，同一个warp中的thread并不是严格同步的，所以要引入__syncwarp保证读操作都能在写操作前完成
    if (blockDim.x >= 64){
        smem[tid] += smem[tid + 32];
        __syncwarp();
    }
    smem[tid] += smem[tid + 16];
    __syncwarp();
    smem[tid] += smem[tid + 8]; 
    __syncwarp();
    smem[tid] += smem[tid + 4]; 
    __syncwarp();
    smem[tid] += smem[tid + 2];
    __syncwarp();
    smem[tid] += smem[tid + 1]; 
    __syncwarp();
}

template<int blockSize>
__global__ void reduce_v4(float* device_in, float* device_out){
    unsigned int tid = threadIdx.x;
    unsigned int gtid = (blockSize * 2) * blockIdx.x + threadIdx.x;

    __shared__ float smem[blockSize];
    smem[tid] = device_in[gtid] + device_in[gtid + blockSize];
    __syncthreads();

    // index = 64
    for (unsigned int index = blockDim.x / 2; index > 32; index >>= 1){
        if (tid < index){
            smem[tid] += smem[tid + index];
        }
        __syncthreads();
    } 

    if (tid < 32){
        warpSharedMemReduce(smem, tid);
    }

    if (tid == 0){
        device_out[blockIdx.x] = smem[0];
    }
}

bool checkResult(float* device_out, float groundtruth, int n){
    float sum = 0;
    for (int i = 0; i < n; i++){
        sum += device_out[i];
    }
    if (sum != groundtruth){
        printf("device_out is %f\n", sum);
        return false;
    }
    return true;
}

int main(){
    const int N = 25600000;

    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    const int blockSize = 256;      // thread数量
    int gridSize = std::min((N + 256 - 1) / 256, deviceProp.maxGridSize[0]);
    dim3 grid(gridSize);
    dim3 block(blockSize / 2);
    
    float* device_in;
    float* host_in = (float* )malloc(N * sizeof(float));
    hipMalloc((void** )&device_in, N * sizeof(float));
    float* device_out;
    float* host_out = (float* )malloc(gridSize * sizeof(float));
    hipMalloc((void** )&device_out, gridSize * sizeof(float));

    for (int i = 0; i < N; ++i){
        host_in[i] = 1;
    }
    
    hipMemcpy(device_in, host_in, N * sizeof(float), hipMemcpyHostToDevice);
    
    float milliseconds = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    reduce_v4<blockSize / 2><<<grid, block>>>(device_in, device_out);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(host_out, device_out, gridSize * sizeof(float), hipMemcpyDeviceToHost);

    float groundtruth = N * 1;
    bool is_right = checkResult(host_out, groundtruth, gridSize);
    if (is_right){
        printf("the ans is right!\n");
    }else{
        printf("groundtruth is %f\n", groundtruth);
        printf("the ans is false\n");
    }
    printf("reduce_v4 latency = %f ms\n", milliseconds);

    hipFree(device_in);
    hipFree(device_out);
    free(host_in);
    free(host_out);

    
    return 0;
}