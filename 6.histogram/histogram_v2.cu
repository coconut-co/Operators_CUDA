// 原子操作（串行）, histogram latency = 1.247872 ms
# include<hip/hip_runtime.h>

# include<iostream>

__global__ void histgram(int* hist_device, int* bin_device, int N){
    __shared__ int smem[256];
    int gtid = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    smem[tid] = 0;  // 初始化shared mem为0
    __syncthreads();

    // 用for循环自动确定 每个线程 处理的元素
    for (int i = gtid; i < N; i += gridDim.x * blockDim.x){
        int val = hist_device[i];   // 每个单线程（gtid）计算全局内存中的若干值
        atomicAdd(&bin_device[val], 1);
    }
    __syncthreads();    // 此时 每 个block负责的数据都已统计在smem中
    atomicAdd(&bin_device[tid], smem[tid]);
}
void checkResult(int* device_out, int* groundtruth, int n){
    for (int i = 0; i < n; i++){
        if (device_out[i] != groundtruth[i]){
            printf("the ans is flase\n");
        }
    }
    printf("the ans is right\n");
}

int main(){
    hipSetDevice(0);
    hipDeviceProp_t deviceprop;
    hipGetDeviceProperties(&deviceprop, 0);
 
    const int N = 25600000;
    const int blockSize = 256;
    int gridSize = std::min((N + 256 - 1) / 256, deviceprop.maxGridSize[0]);    // gridSize=10000
    dim3 block(blockSize);
    dim3 grid(gridSize);

    int* hist = (int* )malloc(N * sizeof(int));
    int* bin = (int* )malloc(256 * sizeof(int));
    int* hist_device;
    int* bin_device;
    hipMalloc((void** )&hist_device, N * sizeof(int));
    hipMalloc((void** )&bin_device, 256 * sizeof(int));

    // 初始化数据
    for (int i = 0; i < N; i++){
        hist[i] = i % 256;  // %让数在0-255之间
    }
    hipMemcpy(hist_device, hist, N * sizeof(int), hipMemcpyHostToDevice);

    float milliseconds = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    histgram<<<grid, block>>>(hist_device, bin_device, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(bin, bin_device, 256 * sizeof(int), hipMemcpyDeviceToHost);

    int *groundtruth = (int* )malloc(256 * sizeof(int));
    for (int i = 0; i < 256; i++){
        groundtruth[i] = 100000;
    }
    checkResult(bin, groundtruth, 256);
    printf("histogram latency = %f ms\n", milliseconds);  

    hipFree(bin_device);
    hipFree(hist_device);
    free(bin);
    free(hist);

}