// 原子操作（串行）, histogram latency = 1.247872 ms
# include<hip/hip_runtime.h>

# include<iostream>

__global__ void histgram(int* hist_device, int* bin_device){
    int gtid = blockDim.x * blockIdx.x + threadIdx.x;
    // 资源竞争：如果线程0和线程1处理的数一样，导致bin_device[1]不知道如何++
    // bin_data[hist_data[gtid]]++;
    // 原子加法，将所有并行线程强制转化为串行
    atomicAdd(&bin_device[hist_device[gtid]], 1);
}
void checkResult(int* device_out, int* groundtruth, int n){
    for (int i = 0; i < n; i++){
        if (device_out[i] != groundtruth[i]){
            printf("the ans is flase\n");
        }
    }
    printf("the ans is right\n");
}

int main(){
    hipSetDevice(0);
    hipDeviceProp_t deviceprop;
    hipGetDeviceProperties(&deviceprop, 0);
 
    const int N = 25600000;
    const int blockSize = 256;
    int gridSize = std::min((N + 256 - 1) / 256, deviceprop.maxGridSize[0]);    // gridSize=10000
    dim3 block(blockSize);
    dim3 grid(gridSize);

    int* hist = (int* )malloc(N * sizeof(int));
    int* bin = (int* )malloc(256 * sizeof(int));
    int* hist_device;
    int* bin_device;
    hipMalloc((void** )&hist_device, N * sizeof(int));
    hipMalloc((void** )&bin_device, 256 * sizeof(int));

    // 初始化数据
    for (int i = 0; i < N; i++){
        hist[i] = i % 256;  // %让数在0-255之间
    }
    hipMemcpy(hist_device, hist, N * sizeof(int), hipMemcpyHostToDevice);

    float milliseconds = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    histgram<<<grid, block>>>(hist_device, bin_device);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(bin, bin_device, 256 * sizeof(int), hipMemcpyDeviceToHost);

    int *groundtruth = (int* )malloc(256 * sizeof(int));
    for (int i = 0; i < 256; i++){
        groundtruth[i] = 100000;
    }
    checkResult(bin, groundtruth, 256);
    printf("histogram latency = %f ms\n", milliseconds);  

    hipFree(bin_device);
    hipFree(hist_device);
    free(bin);
    free(hist);

}